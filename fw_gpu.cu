#include "hip/hip_runtime.h"
/**
PPC- Assignment3
Eduardo Madeira fc51720
*/
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#include "workshop.h"

#define GRAPH_SIZE 2000

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

#define BLOCK_EDGE 16
// funciona para o graphsize 2000, mas nao funciona para qualquer tamanho 
//#define BLOCKS_PER_GRAPH_SIDE GRAPH_SIZE / THREADS_PER_BLOCK_SIDE 
// funciona para qualquer tamanho de graphsize
#define GRID_EDGE ((GRAPH_SIZE+BLOCK_EDGE-1) / BLOCK_EDGE)



void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}


//__global__ keyword means that is a function that can be called on the gpu from the cpu 
__global__ void floyd_warshall_kernel(const int graph_size, int *output, int k) {

  /* errado
  int j = blockIdx.x;
  int i = threadIdx.y;

  while(i< graph_size){
    while(j< graph_size){
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
      }
      j+=blockDim.x;
    }
    i+=gridDim.y;
  }
  */

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
 
  if ((i < graph_size) && (j < graph_size)){
    if (D(i, k) + D(k, j) < D(i, j)) {
      D(i, j) = D(i, k) + D(k, j);
    }
  }

}

void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {
  // TODO
  int *dev_output;

  HANDLE_ERROR(hipMalloc(&dev_output, sizeof(int) * graph_size * graph_size) );
  hipMemcpy(dev_output, graph, sizeof(int) * graph_size * graph_size, hipMemcpyHostToDevice);

  dim3 blocks(GRID_EDGE, GRID_EDGE);
  dim3 threads(BLOCK_EDGE, BLOCK_EDGE);

  int i;
  for (i = 0; i < graph_size; i++) {
    floyd_warshall_kernel<<<blocks, threads>>>(graph_size, dev_output, i);
  }
  
  HANDLE_ERROR(hipMemcpy(output, dev_output, sizeof(int) * graph_size * graph_size, hipMemcpyDeviceToHost));
  hipFree(dev_output);
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  }

  return 0;
}
